#include "hip/hip_runtime.h"
// Copyright (C) 2021-2023 Intel Corporation
// SPDX-License-Identifier: Apache-2.0
//

#include "convert.cuh"
#include "power.hpp"

namespace ov {
namespace nvidia_gpu {
namespace kernel {

template <typename T>
struct PowerOpImpl {
    __device__ static inline T op(T in0, T in1) { return pow(in0, in1); }
};

template <>
struct PowerOpImpl<__half> {
    __device__ static inline __half op(__half in0, __half in1) {
        return cast<__half>(powf(cast<float>(in0), cast<float>(in1)));
    }
};

Power::Power(Type_t element_type, size_t out_num_elements, size_t max_threads_per_block)
    : impl_{element_type, out_num_elements, max_threads_per_block} {}

void Power::operator()(hipStream_t stream,
                       const void* in0,
                       const NumpyBroadcastMapper& in0_mapper,
                       const void* in1,
                       const NumpyBroadcastMapper& in1_mapper,
                       void* out) const {
    impl_(stream, in0, in0_mapper, in1, in1_mapper, out);
}

}  // namespace kernel
}  // namespace nvidia_gpu
}  // namespace ov
