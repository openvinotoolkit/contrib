#include "hip/hip_runtime.h"
// Copyright (C) 2021-2023 Intel Corporation
// SPDX-License-Identifier: Apache-2.0
//

#include <cuda/float16.hpp>

#include "convert.cuh"
#include "details/type_validator.hpp"
#include "details/typed_functor.hpp"
#include "kernels/range.hpp"

namespace ov {
namespace nvidia_gpu {

namespace kernel {

template <typename T_IN1, typename T_IN2, typename T_OUT>
static __global__ typename std::enable_if<std::is_same<T_OUT, __half>::value>::type range(const T_IN1* start,
                                                                                          const T_IN2* step,
                                                                                          const size_t dstSize,
                                                                                          T_OUT* dst) {
    const unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= dstSize) {
        return;
    }
    dst[i] = __hadd(cast<T_OUT, T_IN1>(start[0]),
#ifdef CUDA_HAS_HALF_MATH
                    __hmul(cast<T_OUT, T_IN2>(step[0]), cast<T_OUT, decltype(i)>(i))
#else
                    // the __hmul operation isn't supported here. Also, operation+ and operation* aren't defined here
                    cast<T_OUT, float>(cast<float, T_IN2>(step[0]) * cast<float, decltype(i)>(i))
#endif
    );
}

template <typename T_IN1, typename T_IN2, typename T_OUT>
static __global__ typename std::enable_if<!std::is_same<T_OUT, __half>::value>::type range(const T_IN1* start,
                                                                                           const T_IN2* step,
                                                                                           const size_t dstSize,
                                                                                           T_OUT* dst) {
    const unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= dstSize) {
        return;
    }
    dst[i] = cast<T_OUT>(start[0]) + cast<T_OUT>(step[0]) * cast<T_OUT>(i);
}

template <typename T_IN1, typename T_IN2, typename T_OUT>
struct RangeFunctor {
    static void function(hipStream_t stream,
                         unsigned blocks_number,
                         unsigned threads_per_block,
                         const void* start,
                         const void* step,
                         const size_t dstSize,
                         void* dst) {
        range<T_IN1, T_IN2, T_OUT><<<blocks_number, threads_per_block, 0, stream>>>(
            static_cast<const T_IN1*>(start), static_cast<const T_IN2*>(step), dstSize, static_cast<T_OUT*>(dst));
    }
};

RangeKernelOp::RangeKernelOp(const size_t max_size,
                             const unsigned blocks_number,
                             const unsigned threads_per_block,
                             const Type_t input_start_type,
                             const Type_t input_stop_type,
                             const Type_t input_step_type,
                             const Type_t output_type)
    : blocks_number_{blocks_number}, threads_per_block_{threads_per_block} {
    TypeValidator<AllElementTypesSwitch>::check(input_start_type);
    TypeValidator<AllElementTypesSwitch>::check(input_stop_type);
    TypeValidator<AllElementTypesSwitch>::check(input_step_type);
    TypeValidator<AllElementTypesSwitch>::check(output_type);
    static constexpr TypedFunctor<RangeFunctor, TFuncPtr, DIM_3D> combinations{};
    func_ptr_ = combinations[input_start_type][input_step_type][output_type];
}

void RangeKernelOp::operator()(
    const hipStream_t stream, const void* start, const void* step, const size_t dstSize, void* dst) const {
    func_ptr_(stream, blocks_number_, threads_per_block_, start, step, dstSize, dst);
}

}  // namespace kernel

}  // namespace nvidia_gpu
}  // namespace ov
