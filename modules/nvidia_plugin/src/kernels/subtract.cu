// Copyright (C) 2022-2023 Intel Corporation
// SPDX-License-Identifier: Apache-2.0
//

#include <cuda/float16.hpp>
#include <type_traits>

#include "subtract.hpp"

namespace ov {
namespace nvidia_gpu {
namespace kernel {

template <typename T>
struct SubtractOpImpl {
    __device__ static inline T op(T in0, T in1) { return in0 - in1; }
};

Subtract::Subtract(Type_t element_type, size_t out_num_elements, size_t max_threads_per_block)
    : impl_{element_type, out_num_elements, max_threads_per_block} {}

void Subtract::operator()(hipStream_t stream,
                          const void* in0,
                          const NumpyBroadcastMapper& in0_mapper,
                          const void* in1,
                          const NumpyBroadcastMapper& in1_mapper,
                          void* out) const {
    impl_(stream, in0, in0_mapper, in1, in1_mapper, out);
}

}  // namespace kernel
}  // namespace nvidia_gpu
}  // namespace ov
