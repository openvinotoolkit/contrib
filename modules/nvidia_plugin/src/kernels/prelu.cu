#include "prelu.hpp"

namespace ov {
namespace nvidia_gpu {
namespace kernel {

template <typename T>
struct PReluOpImpl {
    __device__ static inline T op(T in0, T in1) { return in0 > T{} ? in0 : in0 * in1; }
};

PRelu::PRelu(Type_t element_type, size_t out_num_elements, size_t max_threads_per_block)
    : impl_{element_type, out_num_elements, max_threads_per_block} {}

void PRelu::operator()(hipStream_t stream,
                       const void* in0,
                       const NumpyBroadcastMapper& in0_mapper,
                       const void* in1,
                       const NumpyBroadcastMapper& in1_mapper,
                       void* out) const {
    impl_(stream, in0, in0_mapper, in1, in1_mapper, out);
}

}  // namespace kernel
}  // namespace nvidia_gpu
}  // namespace ov
