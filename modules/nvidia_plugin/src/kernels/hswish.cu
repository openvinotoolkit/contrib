#include "hip/hip_runtime.h"
// Copyright (C) 2021-2023 Intel Corporation
// SPDX-License-Identifier: Apache-2.0
//


#include "hswish.hpp"

namespace ov {
namespace nvidia_gpu {
namespace kernel {

namespace cumath = CUDA::math;

template <typename T>
struct HSwishOpImpl {
    __device__ static inline T op(T x) {
        return x * cumath::min(cumath::max(x + static_cast<T>(3.0), static_cast<T>(0.0)),
                               static_cast<T>(6.0)) * static_cast<T>(1.0 / 6.0);
    }
};

HSwish::HSwish(Type_t element_type, size_t max_threads_per_block, size_t num_elements)
    : impl_{element_type, max_threads_per_block, num_elements} {}

void HSwish::operator()(hipStream_t stream, const void* in0, void* out) const {
    impl_(stream, in0, out);
}

}  // namespace kernel
}  // namespace nvidia_gpu
}  // namespace ov
