#include "hip/hip_runtime.h"
// Copyright (C) 2021-2023 Intel Corporation
// SPDX-License-Identifier: Apache-2.0
//

#include "sin.hpp"

namespace ov {
namespace nvidia_gpu {
namespace kernel {

namespace cumath = CUDA::math;

template <typename T>
struct SinOpImpl {
    __device__ static inline T op(T x) {
        return cumath::sin(x);
    }
};

Sin::Sin(Type_t element_type, size_t max_threads_per_block, size_t num_elements)
    : impl_{element_type, max_threads_per_block, num_elements} {}

void Sin::operator()(hipStream_t stream, const void* in0, void* out) const {
    impl_(stream, in0, out);
}

}  // namespace kernel
}  // namespace nvidia_gpu
}  // namespace ov
