// Copyright (C) 2022-2023 Intel Corporation
// SPDX-License-Identifier: Apache-2.0
//

#include "add.hpp"

namespace ov {
namespace nvidia_gpu {
namespace kernel {

template <typename T>
struct AddOpImpl {
    __device__ static inline T op(T in0, T in1) { return in0 + in1; }
};

Add::Add(Type_t element_type, size_t out_num_elements, size_t max_threads_per_block)
    : impl_{element_type, out_num_elements, max_threads_per_block} {}

void Add::operator()(hipStream_t stream,
                     const void* in0,
                     const NumpyBroadcastMapper& in0_mapper,
                     const void* in1,
                     const NumpyBroadcastMapper& in1_mapper,
                     void* out) const {
    impl_(stream, in0, in0_mapper, in1, in1_mapper, out);
}

}  // namespace kernel
}  // namespace nvidia_gpu
}  // namespace ov
