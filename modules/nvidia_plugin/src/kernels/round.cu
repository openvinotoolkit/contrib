#include "hip/hip_runtime.h"
// Copyright (C) 2022-2023 Intel Corporation
// SPDX-License-Identifier: Apache-2.0
//

#include <cuda/float16.hpp>

#include "round.hpp"

namespace ov {
namespace nvidia_gpu {
namespace kernel {

template <typename T>
struct RoundOpImpl {
    __device__ static inline T op(T x) { return CUDA::math::round(x); }
};

Round::Round(Type_t element_type, size_t max_threads_per_block, size_t num_elements)
    : ewu_{element_type, max_threads_per_block, num_elements} {}

void Round::operator()(hipStream_t stream, const void* in, void* out) const { ewu_(stream, in, out); }

}  // namespace kernel
}  // namespace nvidia_gpu
}  // namespace ov
