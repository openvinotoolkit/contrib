#include "hip/hip_runtime.h"
// Copyright (C) 2018-2021 Intel Corporation
// SPDX-License-Identifier: Apache-2.0
//
#include <hip/hip_fp16.h>
#include <fmt/format.h>

#include <cstdint>

#include "details/error.hpp"
#include "details/type_validator.hpp"
#include "pad.cuh"

namespace ov {
namespace nvidia_gpu {
namespace kernel {

// TODO: Would be optimized adding template specializations. Left without this optimization in sake of readability.
template <int PayloadRank>
static inline __device__ bool is_padding(const long ndim_src_indices[PayloadRank],
                                         const std::size_t src_shape[PayloadRank]) {
#pragma unroll PayloadRank
    for (int i = 0; i < PayloadRank; i++) {
        const auto src_dim = ndim_src_indices[i];
        if (src_dim < 0 || src_dim >= src_shape[i]) return true;
    }
    return false;
}

template <int PayloadRank>
static inline __device__ void ndim_dst_indices2ndim_src_indices(const long ndim_dst_indices[PayloadRank],
                                                                const std::size_t pad_begin[PayloadRank],
                                                                long ndim_src_indices[PayloadRank]) {
#pragma unroll PayloadRank
    for (int i = 0; i < PayloadRank; i++) ndim_src_indices[i] = ndim_dst_indices[i] - pad_begin[i];
}

template <int PayloadRank>
static inline __device__ void extract_ndim_dst_indices(long ndim_dst_indices[PayloadRank],
                                                       const std::size_t dst_shape[PayloadRank]) {
#pragma unroll PayloadRank
    for (int i = 0; i < PayloadRank; i++)
        ndim_dst_indices[i] = index_in_dim<PayloadRank>(i, dst_shape, ConstModePad::kElementsPerThread);
}

// TODO: Strides would be precalculated on the host side. Left without this optimization in sake of readability.
template <int PayloadRank>
static inline __device__ std::size_t ndim_indices2index(const long ndim_indices[PayloadRank],
                                                        const std::size_t shape[PayloadRank]) {
    std::size_t stride = 1;
    auto result = ndim_indices[PayloadRank - 1];
#pragma unroll PayloadRank
    for (int i = PayloadRank - 2; i >= 0; i--) {
        stride *= shape[i + 1];
        result += stride * ndim_indices[i];
    }
    return result;
}

template <typename T, int PayloadRank>
static inline __global__ void pad_const_mode(const T* src,
                                             T* dst,
                                             const std::size_t pad_begin[PayloadRank],
                                             const std::size_t src_shape[PayloadRank],
                                             const std::size_t dst_shape[PayloadRank],
                                             const T* pad_value) {
    auto lastDim = PayloadRank - 1;
    auto index_in_last_dim = index_in_dim<PayloadRank>(lastDim, dst_shape, ConstModePad::kElementsPerThread);
    if (index_in_last_dim < dst_shape[lastDim]) {
        const auto dst_index = index<PayloadRank>(dst_shape, ConstModePad::kElementsPerThread);
        long ndim_dst_indices[PayloadRank];
        extract_ndim_dst_indices<PayloadRank>(ndim_dst_indices, dst_shape);
        long ndim_src_indices[PayloadRank];
        ndim_dst_indices2ndim_src_indices<PayloadRank>(ndim_dst_indices, pad_begin, ndim_src_indices);
        if (is_padding<PayloadRank>(ndim_src_indices, src_shape))
            dst[dst_index] = *pad_value;
        else
            dst[dst_index] = src[ndim_indices2index<PayloadRank>(ndim_src_indices, src_shape)];
    }
}

template <typename T>
static inline __global__ void nchw_pad_const_mode(const T* src,
                                                  T* dst,
                                                  const std::size_t pad_begin[4],
                                                  const std::size_t src_shape[4],
                                                  const std::size_t dst_shape[4],
                                                  const T* pad_value,
                                                  const size_t max_element_number) {
    const unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= max_element_number) return;
    enum { N, C, H, W };
    const int output_width = idx % dst_shape[W];
    int nc_index = idx / dst_shape[W];
    const int output_height = nc_index % dst_shape[H];
    nc_index /= dst_shape[H];

    int input_height = output_height - pad_begin[H];
    int input_width = output_width - pad_begin[W];

    dst[idx] = (input_height < 0 || input_width < 0 || input_height >= src_shape[H] || input_width >= src_shape[W])
                   ? *pad_value
                   : src[(nc_index * src_shape[H] + input_height) * src_shape[W] + input_width];
}

ConstModePad::ConstModePad(eltwise::KernelExecAttrs&& kernelExecAttrs,
                           kernel::Type_t dtype,
                           std::size_t outputRank,
                           int maxElementsPerThread,
                           size_t elementsNumber,
                           bool nchw_conv_padding)
    : kernel_exec_attrs_{std::move(kernelExecAttrs)},
      dtype_{dtype},
      output_rank_{outputRank},
      max_elements_per_thread_{maxElementsPerThread},
      elements_number_{elementsNumber},
      nchw_conv_padding_{nchw_conv_padding} {
    using PadElementTypesSwitch = ElementTypesSwitch<Type_t::f32,
                                                     Type_t::i32,
                                                     Type_t::u32,
                                                     Type_t::f16,
                                                     Type_t::i16,
                                                     Type_t::u16,
                                                     Type_t::u8,
                                                     Type_t::i8,
                                                     Type_t::boolean>;
    TypeValidator<PadElementTypesSwitch>::check(dtype);

    const auto elements = static_cast<int>(elements_number_);
    blocks_number_ = 1 + elements / max_elements_per_thread_;
    threads_per_block_ = (blocks_number_ == 1) ? elements : max_elements_per_thread_;
}

void ConstModePad::operator()(hipStream_t stream,
                              const void* src,
                              void* dst,
                              const void* begin,
                              const std::size_t* srcShape,
                              const std::size_t* dstShape,
                              const void* padValue) const {
    /*
     * Since Pad is a data movement operation which doesn't change values,
     * it's type agnostic for types of the same width.
     * In sake of reducing code duplication and binary size, types of the same width are processed
     * by unsigned integer template instantiation version of appropriate width.
     * */
    switch (dtype_) {
        case Type_t::f32:
        case Type_t::i32:
        case Type_t::u32:
            callKernel<std::uint32_t>(stream, src, dst, begin, srcShape, dstShape, padValue);
            break;
        case Type_t::f16:
        case Type_t::i16:
        case Type_t::u16:
            callKernel<std::uint16_t>(stream, src, dst, begin, srcShape, dstShape, padValue);
            break;
        case Type_t::u8:
        case Type_t::i8:
        case Type_t::boolean:
            callKernel<std::uint8_t>(stream, src, dst, begin, srcShape, dstShape, padValue);
            break;
        default:
            throwIEException(fmt::format("Index element type = {} is not supported by Pad operation !", dtype_));
    }
}

template <typename T>
void ConstModePad::callKernel(hipStream_t stream,
                              const void* src,
                              void* dst,
                              const void* begin,
                              const std::size_t* srcShape,
                              const std::size_t* dstShape,
                              const void* padValue) const {
    switch (output_rank_) {
        case 1:
            callKernel<T, 1>(stream, src, dst, begin, srcShape, dstShape, padValue);
            break;
        case 2:
            callKernel<T, 2>(stream, src, dst, begin, srcShape, dstShape, padValue);
            break;
        case 3:
            callKernel<T, 3>(stream, src, dst, begin, srcShape, dstShape, padValue);
            break;
        case 4:
            if (nchw_conv_padding_)
                callNCHWFormatConvKernel<T>(stream, src, dst, begin, srcShape, dstShape, padValue);
            else
                callKernel<T, 4>(stream, src, dst, begin, srcShape, dstShape, padValue);
            break;
        case 5:
            callKernel<T, 5>(stream, src, dst, begin, srcShape, dstShape, padValue);
            break;
    }
}

template <typename T, int PayloadRank>
void ConstModePad::callKernel(hipStream_t stream,
                              const void* src,
                              void* dst,
                              const void* begin,
                              const std::size_t* srcShape,
                              const std::size_t* dstShape,
                              const void* padValue) const {
    pad_const_mode<T, PayloadRank>
        <<<kernel_exec_attrs_.grid, kernel_exec_attrs_.block, 0, stream>>>(static_cast<const T*>(src),
                                                                           static_cast<T*>(dst),
                                                                           static_cast<const size_t*>(begin),
                                                                           srcShape,
                                                                           dstShape,
                                                                           static_cast<const T*>(padValue));
}

template <typename T>
void ConstModePad::callNCHWFormatConvKernel(hipStream_t stream,
                                            const void* src,
                                            void* dst,
                                            const void* begin,
                                            const std::size_t* srcShape,
                                            const std::size_t* dstShape,
                                            const void* padValue) const {
    nchw_pad_const_mode<T><<<blocks_number_, threads_per_block_, 0, stream>>>(static_cast<const T*>(src),
                                                                              static_cast<T*>(dst),
                                                                              static_cast<const size_t*>(begin),
                                                                              srcShape,
                                                                              dstShape,
                                                                              static_cast<const T*>(padValue),
                                                                              elements_number_);
}

}  // namespace kernel
}  // namespace nvidia_gpu
}  // namespace ov
