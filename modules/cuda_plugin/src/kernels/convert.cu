#include "hip/hip_runtime.h"
// Copyright (C) 2018-2021 Intel Corporation
// SPDX-License-Identifier: Apache-2.0
//

#include <array>
#include "convert.cuh"
#include "convert.hpp"
#include "error.hpp"

namespace CUDAPlugin {
namespace kernel {

template <typename TOutput, typename TInput>
__global__
    void convert_impl(size_t inputSize, TOutput* out, const TInput* in) {
    const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < inputSize) {
        out[i] = cast<TOutput>(in[i]);
    }
}

namespace detail {

template <size_t OutputType, size_t InputType>
struct ConvertFunctor {
    static void function(hipStream_t stream,
                         size_t size,
                         void* output,
                         const void* input,
                         unsigned numBlocks,
                         unsigned threadsPerBlock) {
        constexpr Type_t output_type = static_cast<Type_t>(OutputType + type_t_first_value);
        constexpr Type_t input_type = static_cast<Type_t>(InputType + type_t_first_value);
        using TOutput = cuda_type_traits_t<output_type>;
        using TInput = cuda_type_traits_t<input_type>;
        if (OutputType == InputType) {
            if (output == input) return;
            throwIfError(hipMemcpyAsync(output, input, size * sizeof(TOutput), hipMemcpyDeviceToDevice, stream));
        } else {
            CUDAPlugin::kernel::convert_impl<TOutput, TInput><<<numBlocks, threadsPerBlock, 0, stream>>>(
                size, static_cast<TOutput*>(output), static_cast<const TInput*>(input));
        }
    }
};

using convert_t = Convert::convert_t;

constexpr size_t type_count = type_t_last_value - type_t_first_value + 1;

template <template <size_t> class Template>
struct convert_vector : std::array<convert_t, type_count> {
    constexpr convert_vector() : convert_vector(std::make_index_sequence<type_count>()) {}

private:
    template <size_t... I>
    constexpr convert_vector(std::index_sequence<I...>)
        : std::array<convert_t, type_count>{&Template<I>::function...} {}
};

template <template <size_t, size_t> class Template, size_t N>
struct reduce {
    template <size_t M>
    using type = Template<N, M>;
};

template <template <size_t, size_t> class Template>
class convert_matrix : public std::array<std::array<convert_t, type_count>, type_count> {
public:
    constexpr convert_matrix() : convert_matrix<Template>(std::make_index_sequence<type_count>()) {}

private:
    template <size_t... I>
    constexpr convert_matrix(std::index_sequence<I...>)
        : std::array<std::array<convert_t, type_count>, type_count>{
              convert_vector<reduce<Template, I>::template type>{}...} {}
};
}  // namespace detail

Convert::Convert(
    Type_t output_element_type, Type_t input_element_type, size_t size, size_t numBlocks, size_t threadsPerBlock)
    : size_{size}, num_blocks_{numBlocks}, threads_per_block_{threadsPerBlock} {
    convert_kernel_ = getConvertKernel(output_element_type, input_element_type);
}

Convert::convert_t Convert::getConvertKernel(Type_t output_element_type, Type_t input_element_type) {
    static constexpr detail::convert_matrix<detail::ConvertFunctor> matrix{};
    const size_t input_type_index = static_cast<size_t>(input_element_type) - type_t_first_value;
    const size_t output_type_index = static_cast<size_t>(output_element_type) - type_t_first_value;
    return matrix[output_type_index][input_type_index];
}

void Convert::operator()(hipStream_t stream, void* output, const void* src) const {
    convert_kernel_(stream, size_, output, src, num_blocks_, threads_per_block_);
}

}  // namespace kernel
}  // namespace CUDAPlugin
