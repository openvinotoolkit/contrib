#include "hip/hip_runtime.h"
// Copyright (C) 2018-2021 Intel Corporation
// SPDX-License-Identifier: Apache-2.0
//

#include <hip/hip_runtime.h>
#if CUDA_VERSION >= 11000
#include <hip/hip_bf16.h>
#endif
#include <hip/hip_fp16.h>
#include <fmt/format.h>

#include <error.hpp>
#include <gsl/gsl_assert>

#include "split.hpp"

namespace CUDAPlugin {
namespace kernel {

template <typename T>
static __global__ void split(
    const size_t numSplitChunks, const size_t splitStepSize, const size_t numSplits, const T *x, T **y) {
    const unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numSplitChunks) {
        const unsigned splitIdx = i % numSplits;
        const unsigned splitStepIdx = i / numSplits;
        auto src = &x[i * splitStepSize];
        auto dest = &y[splitIdx][splitStepIdx * splitStepSize];
        memcpy(dest, src, sizeof(T) * splitStepSize);
    }
}

Split::Split(Type_t element_type,
             size_t num_splits,
             size_t num_split_chunks,
             size_t split_step_size,
             unsigned num_blocks,
             unsigned threads_per_block)
    : element_type_{element_type},
      num_splits_{num_splits},
      num_split_chunks_{num_split_chunks},
      split_step_size_{split_step_size},
      num_blocks_{num_blocks},
      threads_per_block_{threads_per_block} {}

void Split::operator()(hipStream_t stream, const void *src, void **dst) const {
    switch (element_type_) {
        case Type_t::boolean:
            return Call<bool>(stream, src, dst);
#if CUDA_VERSION >= 11000
        case Type_t::bf16:
            return Call<__hip_bfloat16>(stream, src, dst);
#endif
        case Type_t::f16:
            return Call<__half>(stream, src, dst);
        case Type_t::f32:
            return Call<float>(stream, src, dst);
        case Type_t::f64:
            return Call<double>(stream, src, dst);
        case Type_t::i8:
            return Call<int8_t>(stream, src, dst);
        case Type_t::i16:
            return Call<int16_t>(stream, src, dst);
        case Type_t::i32:
            return Call<int32_t>(stream, src, dst);
        case Type_t::i64:
            return Call<int64_t>(stream, src, dst);
        case Type_t::u8:
            return Call<uint8_t>(stream, src, dst);
        case Type_t::u16:
            return Call<uint16_t>(stream, src, dst);
        case Type_t::u32:
            return Call<uint32_t>(stream, src, dst);
        case Type_t::u64:
            return Call<uint64_t>(stream, src, dst);
        default:
            throwIEException(
                fmt::format("Input element type = {} is not supported by Split operation "
                            "!!",
                            static_cast<Type_t>(element_type_)));
    }
}

template <typename T>
void Split::Call(hipStream_t stream, const void *src, void **dst) const {
    split<T><<<num_blocks_, threads_per_block_, 0, stream>>>(
        num_split_chunks_, split_step_size_, num_splits_, static_cast<const T *>(src), reinterpret_cast<T **>(dst));
}

}  // namespace kernel
}  // namespace CUDAPlugin
