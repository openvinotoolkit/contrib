#include "hip/hip_runtime.h"
// Copyright (C) 2018-2021 Intel Corporation
// SPDX-License-Identifier: Apache-2.0
//

#include <hip/hip_runtime.h>
#if CUDA_VERSION >= 11000
#include <hip/hip_bf16.h>
#endif
#include <hip/hip_fp16.h>
#include <fmt/format.h>

#include <error.hpp>
#include <gsl/gsl_assert>

#include "insert.hpp"
#include "tensor_helpers.hpp"

namespace CUDAPlugin {
namespace kernel {

template <typename T>
static __global__ void insert_part(const Insert::Props* props, const size_t start, const T* x, T* y) {
    const unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t old_rank = rank(props->old_shape);
    const size_t new_rank = rank(props->new_shape);
    assert(old_rank == new_rank);
    Shape<size_t, 5> originalIndexes{};
    shape_indices(props->old_shape, i, originalIndexes);
    Shape<size_t, 5> joinIndexes{};
    memcpy(joinIndexes, originalIndexes, sizeof(originalIndexes));
    joinIndexes[props->axe] = start + joinIndexes[props->axe];
    const size_t flatInputAddress = flat_address(props->new_shape, joinIndexes);
    y[flatInputAddress] = x[i];
}

Insert::Insert(const Type_t element_type, const Props& props, const size_t max_threads_per_block)
    : element_type_{element_type}, props_{props} {
    std::tie(num_blocks_, threads_per_block_) =
        calculateElementwiseGrid(shape_size(props.old_shape), max_threads_per_block);
}

void Insert::operator()(const hipStream_t stream, const void* src, void* dst, const size_t start) const {
    switch (element_type_) {
        case Type_t::boolean:
            return call<bool>(stream, src, dst, start);
#if CUDA_VERSION >= 11000
        case Type_t::bf16:
            return call<__hip_bfloat16>(stream, src, dst, start);
#endif
        case Type_t::f16:
            return call<__half>(stream, src, dst, start);
        case Type_t::f32:
            return call<float>(stream, src, dst, start);
        case Type_t::f64:
            return call<double>(stream, src, dst, start);
        case Type_t::i8:
            return call<int8_t>(stream, src, dst, start);
        case Type_t::i16:
            return call<int16_t>(stream, src, dst, start);
        case Type_t::i32:
            return call<int32_t>(stream, src, dst, start);
        case Type_t::i64:
            return call<int64_t>(stream, src, dst, start);
        case Type_t::u8:
            return call<uint8_t>(stream, src, dst, start);
        case Type_t::u16:
            return call<uint16_t>(stream, src, dst, start);
        case Type_t::u32:
            return call<uint32_t>(stream, src, dst, start);
        case Type_t::u64:
            return call<uint64_t>(stream, src, dst, start);
        default:
            throwIEException(fmt::format("Input element type = {} is not supported by Split operation !!",
                                         static_cast<Type_t>(element_type_)));
    }
}

template <typename T>
void Insert::call(const hipStream_t stream, const void* src, void* dst, const size_t start) const {
    Expects(props_ptr_);
    insert_part<T><<<num_blocks_, threads_per_block_, 0, stream>>>(
        static_cast<const Props*>(props_ptr_), start, static_cast<const T*>(src), static_cast<T*>(dst));
}

}  // namespace kernel
}  // namespace CUDAPlugin
