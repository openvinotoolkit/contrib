#include "hip/hip_runtime.h"
// Copyright (C) 2018-2021 Intel Corporation
// SPDX-License-Identifier: Apache-2.0
//

#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <fmt/format.h>
#if CUDA_VERSION >= 11000
#include <hip/hip_bf16.h>
#endif
#include <hip/hip_fp16.h>

#include <error.hpp>

#include "gather.hpp"

namespace CUDAPlugin {

namespace kernel {

template <typename DataType, typename IndexType>
static inline __device__ void gather(unsigned data_length,
                                     size_t index_range,
                                     unsigned els_per_thread,
                                     unsigned indices_size,
                                     unsigned indices_index,
                                     unsigned dict,
                                     unsigned chunk,
                                     const DataType* src_dict,
                                     const IndexType* src_index,
                                     DataType* dst_data) {
    const auto dict_index = src_index[indices_index];
    if (dict_index >= index_range) {
        // TODO: find a way to handle an error raised in a kernel (assertion or trap) properly
        __trap();
    }
    unsigned thread_offset;
    for (int el = 0; el < els_per_thread; ++el) {
        thread_offset = chunk + el;
        if (thread_offset >= data_length) {
            return;
        }
        dst_data[data_length * (indices_index + dict * indices_size) + thread_offset] =
            src_dict[data_length * (dict_index + dict * index_range) + thread_offset];
    }
}

template <typename DataType, typename IndexType>
static __global__ void chunks_gather(unsigned data_length,
                                     size_t index_range,
                                     unsigned num_dicts,
                                     unsigned dicts_batch_stride,
                                     unsigned indices_batch_stride,
                                     unsigned out_batch_stride,
                                     unsigned els_per_thread,
                                     const DataType* src_dict,
                                     const IndexType* src_index,
                                     DataType* dst_data) {
    const auto indices_size = gridDim.y;
    const auto indices_index = blockIdx.y;
    const auto dict = blockIdx.x % num_dicts;
    const auto batch = blockIdx.x / num_dicts;
    const auto chunk = (blockIdx.z * blockDim.x + threadIdx.x) * els_per_thread;
    gather(data_length,
           index_range,
           els_per_thread,
           indices_size,
           indices_index,
           dict,
           chunk,
           src_dict + batch * dicts_batch_stride,
           src_index + batch * indices_batch_stride,
           dst_data + batch * out_batch_stride);
}

template <typename DataType, typename IndexType>
static __global__ void dicts_gather(unsigned data_length,
                                    size_t index_range,
                                    unsigned num_dicts,
                                    unsigned dicts_batch_stride,
                                    unsigned indices_batch_stride,
                                    unsigned out_batch_stride,
                                    unsigned els_per_thread,
                                    const DataType* src_dict,
                                    const IndexType* src_index,
                                    DataType* dst_data) {
    const auto indices_size = gridDim.y;
    const auto indices_index = blockIdx.y;
    const auto dict = blockIdx.z * blockDim.x + threadIdx.x;
    if (dict >= num_dicts) {
        return;
    }
    const auto chunk = blockIdx.x % data_length * els_per_thread;
    const auto batch = blockIdx.x / data_length;
    gather(data_length,
           index_range,
           els_per_thread,
           indices_size,
           indices_index,
           dict,
           chunk,
           src_dict + batch * dicts_batch_stride,
           src_index + batch * indices_batch_stride,
           dst_data + batch * out_batch_stride);
}

Gather::Gather(Type_t element_type,
               Type_t indices_type,
               unsigned num_dicts,
               unsigned index_range,
               unsigned data_length,
               unsigned indices_size,
               bool gather_chunks,
               unsigned blocks_per_grid,
               unsigned threads_per_block,
               unsigned grid_dim_x,
               unsigned dicts_batch_stride,
               unsigned indices_batch_stride,
               unsigned out_batch_stride,
               unsigned els_per_thread_chunks,
               unsigned els_per_thread_dicts)
    : element_type_(element_type),
      indices_type_(indices_type),
      num_dicts_(num_dicts),
      index_range_(index_range),
      data_length_(data_length),
      indices_size_(indices_size),
      gather_chunks_(gather_chunks),
      blocks_per_grid_(blocks_per_grid),
      threads_per_block_(threads_per_block),
      grid_dim_x_(grid_dim_x),
      dicts_batch_stride_(dicts_batch_stride),
      indices_batch_stride_(indices_batch_stride),
      out_batch_stride_(out_batch_stride),
      els_per_thread_chunks_(els_per_thread_chunks),
      els_per_thread_dicts_(els_per_thread_dicts) {}

void Gather::operator()(const hipStream_t stream, const void* src_dict, const void* src_index, void* dst_data) const {
    switch (indices_type_) {
        case Type_t::i64:
            return CallByDataType<int64_t>(stream, src_dict, src_index, dst_data);
        case Type_t::i32:
            return CallByDataType<int32_t>(stream, src_dict, src_index, dst_data);
        default:
            throwIEException(
                fmt::format("Index element type = {} is not supported by Gather operation !!", indices_type_));
    }
}

template <typename IndexType>
void Gather::CallByDataType(const hipStream_t stream,
                            const void* src_dict,
                            const void* src_index,
                            void* dst_data) const {
    switch (element_type_) {
        case Type_t::boolean:
            return Call<bool, IndexType>(stream, src_dict, src_index, dst_data);
#if CUDA_VERSION >= 11000
        case Type_t::bf16:
            return Call<__hip_bfloat16, IndexType>(stream, src_dict, src_index, dst_data);
#endif
        case Type_t::f16:
            return Call<__half, IndexType>(stream, src_dict, src_index, dst_data);
        case Type_t::f32:
            return Call<float, IndexType>(stream, src_dict, src_index, dst_data);
        case Type_t::f64:
            return Call<double, IndexType>(stream, src_dict, src_index, dst_data);
        case Type_t::i8:
            return Call<int8_t, IndexType>(stream, src_dict, src_index, dst_data);
        case Type_t::i16:
            return Call<int16_t, IndexType>(stream, src_dict, src_index, dst_data);
        case Type_t::i32:
            return Call<int32_t, IndexType>(stream, src_dict, src_index, dst_data);
        case Type_t::i64:
            return Call<int64_t, IndexType>(stream, src_dict, src_index, dst_data);
        case Type_t::u8:
            return Call<uint8_t, IndexType>(stream, src_dict, src_index, dst_data);
        case Type_t::u16:
            return Call<uint16_t, IndexType>(stream, src_dict, src_index, dst_data);
        case Type_t::u32:
            return Call<uint32_t, IndexType>(stream, src_dict, src_index, dst_data);
        case Type_t::u64:
            return Call<uint64_t, IndexType>(stream, src_dict, src_index, dst_data);
        default:
            throwIEException(
                fmt::format("Index element type = {} is not supported by Gather operation !!", indices_type_));
    }
}

template <typename DataType, typename IndexType>
void Gather::Call(const hipStream_t stream, const void* src_dict, const void* src_index, void* dst_data) const {
    dim3 grid{grid_dim_x_, indices_size_, blocks_per_grid_};

    const auto src_dict_typed = static_cast<const DataType*>(src_dict);
    const auto src_index_typed = static_cast<const IndexType*>(src_index);
    auto dst_data_typed = static_cast<DataType*>(dst_data);

    if (gather_chunks_) {
        kernel::chunks_gather<<<grid, threads_per_block_, 0, stream>>>(data_length_,
                                                                       index_range_,
                                                                       num_dicts_,
                                                                       dicts_batch_stride_,
                                                                       indices_batch_stride_,
                                                                       out_batch_stride_,
                                                                       els_per_thread_chunks_,
                                                                       src_dict_typed,
                                                                       src_index_typed,
                                                                       dst_data_typed);
    } else {
        kernel::dicts_gather<<<grid, threads_per_block_, 0, stream>>>(data_length_,
                                                                      index_range_,
                                                                      num_dicts_,
                                                                      dicts_batch_stride_,
                                                                      indices_batch_stride_,
                                                                      els_per_thread_dicts_,
                                                                      out_batch_stride_,
                                                                      src_dict_typed,
                                                                      src_index_typed,
                                                                      dst_data_typed);
    }
    // TODO: find a way to handle an error raised in a kernel (assertion or trap) properly in CUDA Plugin
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        throwIEException(hipGetErrorString(err));
    }
}

}  // namespace kernel
}  // namespace CUDAPlugin
