#include "hip/hip_runtime.h"
// Copyright (C) 2022 Intel Corporation
// SPDX-License-Identifier: Apache-2.0
//

#include <cuda/float16.hpp>

#include "floor.hpp"

namespace CUDAPlugin {
namespace kernel {

template <typename T>
struct FloorOpImpl {
    __device__ static inline T op(T x) { return ::floor(x); }
};

template <>
struct FloorOpImpl<float> {
    __device__ static inline float op(float x) { return ::floorf(x); }
};

template <>
struct FloorOpImpl<__half> {
    __device__ static inline __half op(__half x) {
#ifdef CUDA_HAS_HALF_MATH
        return ::hfloor(x);
#else
        return FloorOpImpl<float>::op(static_cast<float>(x));
#endif  // CUDA_HAS_HALF_MATH
    }
};

#ifdef CUDA_HAS_BF16_TYPE
template <>
struct FloorOpImpl<__hip_bfloat16> {
    __device__ static inline __hip_bfloat16 op(__hip_bfloat16 x) {
#ifdef CUDA_HAS_BF16_MATH
        return ::hfloor(x);
#else
        return FloorOpImpl<float>::op(static_cast<float>(x));
#endif  // CUDA_HAS_BF16_MATH
    }
};
#endif  // CUDA_HAS_BF16_TYPE

Floor::Floor(Type_t element_type, size_t max_threads_per_block, size_t num_elements)
    : ewu_{element_type, max_threads_per_block, num_elements} {}

void Floor::operator()(hipStream_t stream, const void* in, void* out) const { ewu_(stream, in, out); }

}  // namespace kernel
}  // namespace CUDAPlugin
