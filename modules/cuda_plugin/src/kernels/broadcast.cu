#include "hip/hip_runtime.h"
// Copyright (C) 2021 Intel Corporation
// SPDX-License-Identifier: Apache-2.0
//

#include <fmt/format.h>

#include "broadcast.hpp"
#include "elementtypeswitch.hpp"
#include "tensor_helpers.hpp"

namespace CUDAPlugin {
namespace kernel {

template <typename T>
static __global__ void broadcast(const T* src,
                                 T* dst,
                                 size_t rank,
                                 const size_t* broadcast_dims,
                                 const size_t* src_strides,
                                 const size_t* dst_strides,
                                 size_t dst_num_elements) {
    const unsigned dst_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (dst_idx >= dst_num_elements) {
        return;
    }

    unsigned src_idx = 0;
    unsigned i = dst_idx;
    for (unsigned r = 0; r < rank; r++) {
        unsigned dst_stride = dst_strides[r];
        unsigned dst_coord = i / dst_stride;
        i = i % dst_stride;
        unsigned src_coord = broadcast_dims[r] * dst_coord;
        src_idx += src_coord * src_strides[r];
    }

    dst[dst_idx] = src[src_idx];
}

Broadcast::Broadcast(CUDAPlugin::kernel::Type_t element_type,
                     size_t shape_rank,
                     size_t dst_num_elements,
                     size_t max_threads_per_block)
    : element_type_{element_type}, shape_rank_{shape_rank}, dst_num_elements_{dst_num_elements} {
    std::tie(num_blocks_, threads_per_block_) = calculateElementwiseGrid(dst_num_elements_, max_threads_per_block);
}

void Broadcast::operator()(const hipStream_t stream,
                           const void* src,
                           void* dst,
                           const size_t* broadcast_dims,
                           const size_t* src_strides,
                           const size_t* dst_strides) const {
    AllElementTypesSwitch::switch_(element_type_, *this, stream, src, dst, broadcast_dims, src_strides, dst_strides);
}

template <typename T, typename... Args>
constexpr void Broadcast::case_(hipStream_t stream, Args&&... args) const noexcept {
    callKernel<T>(stream, std::forward<Args>(args)...);
}

template <typename T, typename... Args>
void Broadcast::default_(T t, hipStream_t, const void*, void*, Args...) const noexcept {
    throwIEException(fmt::format("Element type = {} is not supported by Broadcast operation.", t));
}

template <typename T>
void Broadcast::callKernel(const hipStream_t stream,
                           const void* src,
                           void* dst,
                           const size_t* broadcast_dims,
                           const size_t* src_strides,
                           const size_t* dst_strides) const {
    broadcast<T><<<num_blocks_, threads_per_block_, 0, stream>>>(static_cast<const T*>(src),
                                                                 static_cast<T*>(dst),
                                                                 shape_rank_,
                                                                 broadcast_dims,
                                                                 src_strides,
                                                                 dst_strides,
                                                                 dst_num_elements_);
}

}  // namespace kernel
}  // namespace CUDAPlugin
