#include "hip/hip_runtime.h"
// Copyright (C) 2018-2021 Intel Corporation
// SPDX-License-Identifier: Apache-2.0
//

#include <vector>
#include <hip/hip_runtime.h>
#include <gsl/gsl_assert>
#include <cuda/device.hpp>
#include <cuda_operation_registry.hpp>
#include <utility>
#include <fmt/format.h>

#include "cuda/runtime.hpp"
#include "details/cuda_ngraph_import.hpp"
#include "split.hpp"

namespace CUDAPlugin {

template <typename T>
static __global__ void split(const size_t numSplitChunks,
                             const size_t splitStepSize,
                             const size_t numSplits,
                             const T *x,
                             T **y) {
    const unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numSplitChunks) {
        const unsigned splitIdx = i % numSplits;
        const unsigned splitStepIdx = i / numSplits;
        auto src = &x[i*splitStepSize];
        auto dest = &y[splitIdx][splitStepIdx * splitStepSize];
        memcpy(dest, src, sizeof(T) * splitStepSize);
    }
}

SplitOp::SplitOp(const CUDA::CreationContext& context,
                 const ngraph::Node& node,
                 IndexCollection&& inputIds,
                 IndexCollection&& outputIds)
    : OperationBase(context, node, std::move(inputIds), std::move(outputIds)) {
    auto splitOp = dynamic_cast<const ngraph::op::v1::Split*>(&node);
    Expects(splitOp);
    auto input_element_type = splitOp->get_input_element_type(0);
    auto axisNode = dynamic_cast<ngraph::op::v0::Constant*>(splitOp->get_input_node_ptr(1));
    Expects(axisNode);
    auto output_element_type = splitOp->get_output_element_type(0);
    Expects(splitOp->get_input_size() == 2);
    num_splits_ = splitOp->get_num_splits();
    Ensures(num_splits_ != 0);
    Expects(splitOp->get_output_size() == num_splits_);
    Expects(input_element_type == output_element_type);
    switch (input_element_type) {
        case ngraph::element::Type_t::undefined:
        case ngraph::element::Type_t::dynamic:
        case ngraph::element::Type_t::u1:
            THROW_IE_EXCEPTION << fmt::format("Input element type = {} is not supported by Split operation !!",
                                              static_cast<ngraph::element::Type_t>(input_element_type));
    }
    element_type_ = input_element_type;

    auto& dataShape = splitOp->get_input_shape(0);
    const int64_t axis = *axisNode->get_data_ptr<int64_t>();
    Expects(axis >= 0 && axis < dataShape.size());
    Expects(dataShape[axis] % num_splits_ == 0);
    split_step_size_ = (dataShape[axis] / num_splits_) * std::accumulate(dataShape.begin()+axis+1, dataShape.end(), 1, std::multiplies<size_t>());
    Ensures(split_step_size_ != 0);
    num_split_chunks_ = std::accumulate(dataShape.begin(), dataShape.end(), 1, std::multiplies<size_t>()) / split_step_size_;
    Ensures(num_split_chunks_ != 0);
}

WorkbufferRequest SplitOp::GetWorkBufferRequest() const {
  return { {}, { mutableWbSize() } };
}

void SplitOp::Execute(const InferenceRequestContext& context, Inputs inputs, Outputs outputs, const Workbuffers& buffers) {
    switch (element_type_) {
        case ngraph::element::Type_t::boolean: return Execute<bool>(context, inputs, outputs, buffers);
        case ngraph::element::Type_t::bf16: return Execute<__hip_bfloat16>(context, inputs, outputs, buffers);
        case ngraph::element::Type_t::f16: return Execute<__half>(context, inputs, outputs, buffers);
        case ngraph::element::Type_t::f32: return Execute<float>(context, inputs, outputs, buffers);
        case ngraph::element::Type_t::f64: return Execute<double>(context, inputs, outputs, buffers);
        case ngraph::element::Type_t::i8: return Execute<int8_t>(context, inputs, outputs, buffers);
        case ngraph::element::Type_t::i16: return Execute<int16_t>(context, inputs, outputs, buffers);
        case ngraph::element::Type_t::i32: return Execute<int32_t>(context, inputs, outputs, buffers);
        case ngraph::element::Type_t::i64: return Execute<int64_t>(context, inputs, outputs, buffers);
        case ngraph::element::Type_t::u8: return Execute<uint8_t>(context, inputs, outputs, buffers);
        case ngraph::element::Type_t::u16: return Execute<uint16_t>(context, inputs, outputs, buffers);
        case ngraph::element::Type_t::u32: return Execute<uint32_t>(context, inputs, outputs, buffers);
        case ngraph::element::Type_t::u64: return Execute<uint64_t>(context, inputs, outputs, buffers);
        default: THROW_IE_EXCEPTION << fmt::format("Input element type = {} is not supported by Split operation !!",
                                                   static_cast<ngraph::element::Type_t>(element_type_));
    }
}

template <typename T>
void SplitOp::Execute(const InferenceRequestContext& context, Inputs inputs, Outputs outputs, const Workbuffers& buffers) {
    Expects(inputs.size() == 2);
    Expects(outputs.size() == num_splits_);
    Expects(buffers.mutable_buffers.size() == 1);
    auto& threadContext = context.getThreadContext();
    auto& stream = threadContext.stream();
    const unsigned maxBlockSize = threadContext.device().props().maxThreadsPerBlock;
    const unsigned numBlocks = (num_split_chunks_ % maxBlockSize == 0) ?
                               (num_split_chunks_ / maxBlockSize) :
                               (num_split_chunks_ / maxBlockSize + 1);
    const unsigned threadsPerBlock = (numBlocks == 1) ? num_split_chunks_ : maxBlockSize;
    auto outputPtrs = buffers.mutable_buffers[0];
    stream.upload(outputPtrs.get(), reinterpret_cast<T **>(outputs.data()), sizeof(T *) * num_splits_);
    auto in = inputs[0];
    split<T><<<numBlocks, threadsPerBlock, 0, stream.get()>>>(
        num_split_chunks_,
        split_step_size_,
        num_splits_,
        static_cast<const T *>(in.get()),
        reinterpret_cast<T **>(outputPtrs.get()));
}

OPERATION_REGISTER(SplitOp, Split);
} // namespace CUDAPlugin
