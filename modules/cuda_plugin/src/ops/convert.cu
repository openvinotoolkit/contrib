#include "hip/hip_runtime.h"
// Copyright (C) 2018-2021 Intel Corporation
// SPDX-License-Identifier: Apache-2.0
//

#include <array>
#include <hip/hip_runtime.h>
#include <gsl/gsl_assert>
#include <utility>
#include "details/cuda_ngraph_import.hpp"
#include <cuda_operation_registry.hpp>
#include <cuda/device.hpp>
#include <gpu/device_pointers.hpp>

#include <cuda/cuda_type_traits.hpp>
#include "convert.hpp"

namespace CUDAPlugin {

namespace kernel {

template<typename TOutput, typename TInput>
__global__ void convert_impl(size_t inputSize, TOutput * out, const TInput *in) {
  const size_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < inputSize) {
    if constexpr(std::is_same_v<TInput, __half> || std::is_same_v<TInput, __hip_bfloat16> ||
                 std::is_same_v<TOutput, __half> || std::is_same_v<TOutput, __hip_bfloat16>) {
      // workaround for "error: more than one conversion function from "const __half" to "..." applies"
      // converting __half, __hip_bfloat16 via float
      out[i] = static_cast<TOutput>(static_cast<float>(in[i]));
    } else {
      out[i] = static_cast<TOutput>(in[i]);
    }
  }
}
} //namespace kernel

ConvertOp::ConvertOp(const CUDA::Device& device,
                     const std::shared_ptr<ngraph::Node>& node,
                     IndexCollection&& inputIds,
                     IndexCollection&& outputIds)
    : OperationBase(device, node, std::move(inputIds), std::move(outputIds)) {
    Type_t input_element_type = node->get_input_element_type(0);
    Type_t output_element_type = node->get_output_element_type(0);
    Expects(input_element_type >= Type_t::boolean && input_element_type <= Type_t::u64);
    Expects(output_element_type >= Type_t::boolean && output_element_type <= Type_t::u64);
    if (input_element_type == Type_t::u1 || output_element_type == Type_t::u1)
      THROW_IE_EXCEPTION << "Unsupported data type : " << Type_t::u1;
    auto input_shape = node->get_input_shape(0);
    auto output_shape = node->get_output_shape(0);
    size_ = std::accumulate(input_shape.begin(), input_shape.end(), 1, std::multiplies<size_t>());
    auto output_size_ = std::accumulate(output_shape.begin(), output_shape.end(), 1, std::multiplies<size_t>());
    Expects(size_ == output_size_);
    convert_kernel_ = getConvertKernel(output_element_type, input_element_type);
}

void ConvertOp::Execute(const InferenceRequestContext& context, Inputs inputs, Outputs outputs, const Workbuffers&) {
    Expects(inputs.size() == 1);
    Expects(outputs.size() == 1);
    const auto& stream = context.getThreadContext().stream();
    const unsigned maxBlockSize = CudaDevice::GetMaxGridBlockSizeParams(context.getThreadContext().device().currentId());
    const unsigned numBlocks = (size_ % maxBlockSize == 0) ?
                               (size_ / maxBlockSize) :
                               (size_ / maxBlockSize + 1);
    const unsigned threadsPerBlock = (numBlocks == 1) ? size_ : maxBlockSize;
    convert_kernel_(stream, size_, outputs[0], inputs[0], numBlocks, threadsPerBlock);
}

OPERATION_REGISTER(ConvertOp, Convert);

namespace detail {
using namespace kernel;

template<size_t OutputType, size_t InputType>
struct Convert {
  static void function(const CUDA::Stream& stream, size_t size,
                       InferenceEngine::gpu::DevicePointer<void*> output,
                       InferenceEngine::gpu::DevicePointer<const void*> input,
                       unsigned numBlocks, unsigned threadsPerBlock) {
    using namespace InferenceEngine::gpu;
    using namespace ngraph;
    using namespace ngraph::element;
    constexpr Type_t output_type =  static_cast<Type_t>(OutputType + static_cast<size_t>(Type_t::boolean));
    constexpr Type_t input_type =  static_cast<Type_t>(InputType + static_cast<size_t>(Type_t::boolean));
    using TOutput = typename cuda_type_traits<output_type>::value_type;
    using TInput = typename cuda_type_traits<input_type>::value_type;
    if (OutputType == InputType) {
      if (output.get() == input.get()) return;
      throwIfError(hipMemcpyAsync(output.get(), input.get(),
                                   size * sizeof(TOutput),
                                   hipMemcpyDeviceToDevice, stream.get()));
    } else {
      convert_impl<TOutput, TInput><<<numBlocks, threadsPerBlock, 0, stream.get()>>>(
          size, static_cast<TOutput *>(output.get()), static_cast<const TInput *>(input.get()));
    }
  }
};

using Type_t = ngraph::element::Type_t;
using convert_t = ConvertOp::convert_t;

constexpr size_t type_count = static_cast<size_t>(Type_t::u64) - static_cast<size_t>(Type_t::boolean) + 1;

template<template<size_t> class Template>
struct convert_vector : std::array<convert_t, type_count> {
  constexpr convert_vector() : convert_vector(std::make_index_sequence<type_count>()) {}
private:
  template<size_t ... I>
  constexpr convert_vector(std::index_sequence<I...>) : std::array<convert_t, type_count> { &Template<I>::function ... } {}
};

template<template<size_t, size_t> class Template, size_t N>
struct reduce {
  template<size_t M>
  using type = Template<N, M>;
};

template<template<size_t, size_t> class Template>
class convert_matrix : public std::array<std::array<convert_t, type_count>, type_count> {
public:
  constexpr convert_matrix() : convert_matrix<Template>(std::make_index_sequence<type_count>()) {}
private:
  template<size_t ... I>
  constexpr convert_matrix(std::index_sequence<I...>) : std::array<std::array<convert_t, type_count>, type_count> {
    convert_vector<reduce<Template, I>::template type>{} ... } {}
};
} //namespace detail

ConvertOp::convert_t ConvertOp::getConvertKernel(Type_t output_element_type, Type_t input_element_type) {
  static constexpr detail::convert_matrix<detail::Convert> matrix {};
  const size_t input_type_index = static_cast<size_t>(input_element_type) - static_cast<size_t>(Type_t::boolean);
  const size_t output_type_index = static_cast<size_t>(output_element_type) - static_cast<size_t>(Type_t::boolean);
  return matrix[output_type_index][input_type_index];
}
} // namespace CUDAPlugin

